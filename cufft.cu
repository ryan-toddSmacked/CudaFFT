#include "hip/hip_runtime.h"
#include "cufft.cuh"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <>

#include <stdio.h>
#include <stdint.h>		// uintptr_t


// ifft2 scale kernel. Divide each element by the number of elements in the array
static __global__ void ifft2_scale_kernel(hipFloatComplex* d_data, int width, int height, double scale, int pitch)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	size_t idx = 0;
	if (x < width && y < height)
	{
		idx = (size_t)y * pitch + x;
		d_data[idx].x *= scale;
		d_data[idx].y *= scale;
	}
}

// ifft2 scale kernel. Divide each element by the number of elements in the array
static __global__ void ifft2_scale_kernel(hipDoubleComplex* d_data, int width, int height, double scale, int pitch)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	size_t idx = 0;
	if (x < width && y < height)
	{
		idx = (size_t)y * pitch + x;
		d_data[idx].x *= scale;
		d_data[idx].y *= scale;
	}
}

// ifft scale kernel. Divide each element by the number of elements in the array
static __global__ void ifft_scale_kernel(hipFloatComplex* d_data, int N, double scale)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < N)
	{
		d_data[idx].x *= scale;
		d_data[idx].y *= scale;
	}
}

// ifft scale kernel. Divide each element by the number of elements in the array
static __global__ void ifft_scale_kernel(hipDoubleComplex* d_data, int N, double scale)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < N)
	{
		d_data[idx].x *= scale;
		d_data[idx].y *= scale;
	}
}

// 1D fft shift kernel
static __global__ void fftshift_even_kernel(const hipFloatComplex* d_in, hipFloatComplex* d_out, int N)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < N)
	{
		// Pointers are gaurenteed to be different, so no need for a temporary variable
		// N is gaurenteed to be even

		int idx2 = (idx + N / 2) % N;
		d_out[idx] = d_in[idx2];
	}
}

// 1D fft shift kernel
static __global__ void fftshift_odd_kernel(const hipFloatComplex* d_in, hipFloatComplex* d_out, int N)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int half_N = N / 2;
	if (idx < N)
	{
		if (idx <= half_N)
		{
			int idx2 = idx + half_N;
			d_out[idx2] = d_in[idx];
		}
		else
		{
			int idx2 = idx - half_N - 1;
			d_out[idx2] = d_in[idx];
		}
	}
}

// 1D fft shift kernel
static __global__ void fftshift_even_kernel(const hipDoubleComplex* d_in, hipDoubleComplex* d_out, int N)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < N)
	{
		// Pointers are gaurenteed to be different, so no need for a temporary variable
		// N is gaurenteed to be even

		int idx2 = (idx + N / 2) % N;
		d_out[idx] = d_in[idx2];
	}
}

// 1D fft shift kernel
static __global__ void fftshift_odd_kernel(const hipDoubleComplex* d_in, hipDoubleComplex* d_out, int N)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int half_N = N / 2;
	if (idx < N)
	{
		if (idx <= half_N)
		{
			int idx2 = idx + half_N;
			d_out[idx2] = d_in[idx];
		}
		else
		{
			int idx2 = idx - half_N - 1;
			d_out[idx2] = d_in[idx];
		}
	}
}

static __global__ void fftshift_even_kernel_width(const hipFloatComplex* d_in, hipFloatComplex* d_out, int width, int height, int in_pitch, int out_pitch)
{
	// Only shifting along the width, y will remain the same

	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int half_width = width / 2;

	if (x < width && y < height)
	{
		int x2 = (x + half_width) % width;
		size_t idx = (size_t)y * out_pitch + x;
		size_t idx2 = (size_t)y * in_pitch + x2;
		d_out[idx] = d_in[idx2];
	}
}

static __global__ void fftshift_odd_kernel_width(const hipFloatComplex* d_in, hipFloatComplex* d_out, int width, int height, int in_pitch, int out_pitch)
{
	// Only shifting along the width, y will remain the same
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int half_width = width / 2;
	if (x < width && y < height)
	{
		if (x <= half_width)
		{
			int x2 = x + half_width;
			size_t idx = (size_t)y * out_pitch + x2;
			size_t idx2 = (size_t)y * in_pitch + x;
			d_out[idx] = d_in[idx2];
		}
		else
		{
			int x2 = x - half_width - 1;
			size_t idx = (size_t)y * out_pitch + x2;
			size_t idx2 = (size_t)y * in_pitch + x;
			d_out[idx] = d_in[idx2];
		}
	}
}

static __global__ void fftshift_even_kernel_width(const hipDoubleComplex* d_in, hipDoubleComplex* d_out, int width, int height, int in_pitch, int out_pitch)
{
	// Only shifting along the width, y will remain the same

	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int half_width = width / 2;

	if (x < width && y < height)
	{
		int x2 = (x + half_width) % width;
		size_t idx = (size_t)y * out_pitch + x;
		size_t idx2 = (size_t)y * in_pitch + x2;
		d_out[idx] = d_in[idx2];
	}
}

static __global__ void fftshift_odd_kernel_width(const hipDoubleComplex* d_in, hipDoubleComplex* d_out, int width, int height, int in_pitch, int out_pitch)
{
	// Only shifting along the width, y will remain the same
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int half_width = width / 2;
	if (x < width && y < height)
	{
		if (x <= half_width)
		{
			int x2 = x + half_width;
			size_t idx = (size_t)y * out_pitch + x2;
			size_t idx2 = (size_t)y * in_pitch + x;
			d_out[idx] = d_in[idx2];
		}
		else
		{
			int x2 = x - half_width - 1;
			size_t idx = (size_t)y * out_pitch + x2;
			size_t idx2 = (size_t)y * in_pitch + x;
			d_out[idx] = d_in[idx2];
		}
	}
}


static __global__ void fftshift_even_kernel_height(const hipFloatComplex* d_in, hipFloatComplex* d_out, int width, int height, int in_pitch, int out_pitch)
{
	// Only shifting along the height, x will remain the same
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int half_height = height / 2;
	if (x < width && y < height)
	{
		int y2 = (y + half_height) % height;
		size_t idx = (size_t)y * out_pitch + x;
		size_t idx2 = (size_t)y2 * in_pitch + x;
		d_out[idx] = d_in[idx2];
	}
}

static __global__ void fftshift_odd_kernel_height(const hipFloatComplex* d_in, hipFloatComplex* d_out, int width, int height, int in_pitch, int out_pitch)
{
	// Only shifting along the height, x will remain the same
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int half_height = height / 2;
	if (x < width && y < height)
	{
		if (y <= half_height)
		{
			int y2 = y + half_height;
			size_t idx = (size_t)y2 * out_pitch + x;
			size_t idx2 = (size_t)y * in_pitch + x;
			d_out[idx] = d_in[idx2];
		}
		else
		{
			int y2 = y - half_height - 1;
			size_t idx = (size_t)y2 * out_pitch + x;
			size_t idx2 = (size_t)y * in_pitch + x;
			d_out[idx] = d_in[idx2];
		}
	}
}

static __global__ void fftshift_even_kernel_height(const hipDoubleComplex* d_in, hipDoubleComplex* d_out, int width, int height, int in_pitch, int out_pitch)
{
	// Only shifting along the height, x will remain the same
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int half_height = height / 2;
	if (x < width && y < height)
	{
		int y2 = (y + half_height) % height;
		size_t idx = (size_t)y * out_pitch + x;
		size_t idx2 = (size_t)y2 * in_pitch + x;
		d_out[idx] = d_in[idx2];
	}
}

static __global__ void fftshift_odd_kernel_height(const hipDoubleComplex* d_in, hipDoubleComplex* d_out, int width, int height, int in_pitch, int out_pitch)
{
	// Only shifting along the height, x will remain the same
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int half_height = height / 2;
	if (x < width && y < height)
	{
		if (y <= half_height)
		{
			int y2 = y + half_height;
			size_t idx = (size_t)y2 * out_pitch + x;
			size_t idx2 = (size_t)y * in_pitch + x;
			d_out[idx] = d_in[idx2];
		}
		else
		{
			int y2 = y - half_height - 1;
			size_t idx = (size_t)y2 * out_pitch + x;
			size_t idx2 = (size_t)y * in_pitch + x;
			d_out[idx] = d_in[idx2];
		}
	}
}

static __global__ void fftshift2d_even_width_even_height(const hipFloatComplex* d_in, hipFloatComplex* d_out, int width, int height, int in_pitch, int out_pitch)
{
	// Shifting along both the width and height
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int half_width = width / 2;
	int half_height = height / 2;
	if (x < width && y < height)
	{
		int x2 = (x + half_width) % width;
		int y2 = (y + half_height) % height;
		size_t idx = (size_t)y * out_pitch + x;
		size_t idx2 = (size_t)y2 * in_pitch + x2;
		d_out[idx] = d_in[idx2];
	}
}

static __global__ void fftshift2d_odd_width_even_height(const hipFloatComplex* d_in, hipFloatComplex* d_out, int width, int height, int in_pitch, int out_pitch)
{
	// Shifting along both the width and height
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int half_width = width / 2;
	int half_height = height / 2;
	if (x < width && y < height)
	{
		if (x <= half_width)
		{
			int x2 = x + half_width;
			int y2 = (y + half_height) % height;
			size_t idx = (size_t)y2 * out_pitch + x2;
			size_t idx2 = (size_t)y * in_pitch + x;
			d_out[idx] = d_in[idx2];
		}
		else
		{
			int x2 = x - half_width - 1;
			int y2 = (y + half_height) % height;
			size_t idx = (size_t)y2 * out_pitch + x2;
			size_t idx2 = (size_t)y * in_pitch + x;
			d_out[idx] = d_in[idx2];
		}
	}
}

static __global__ void fftshift2d_even_width_odd_height(const hipFloatComplex* d_in, hipFloatComplex* d_out, int width, int height, int in_pitch, int out_pitch)
{
	// Shifting along both the width and height
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int half_width = width / 2;
	int half_height = height / 2;
	if (x < width && y < height)
	{
		if (y <= half_height)
		{
			int x2 = (x + half_width) % width;
			int y2 = y + half_height;
			size_t idx = (size_t)y2 * out_pitch + x2;
			size_t idx2 = (size_t)y * in_pitch + x;
			d_out[idx] = d_in[idx2];
		}
		else
		{
			int x2 = (x + half_width) % width;
			int y2 = y - half_height - 1;
			size_t idx = (size_t)y2 * out_pitch + x2;
			size_t idx2 = (size_t)y * in_pitch + x;
			d_out[idx] = d_in[idx2];
		}
	}
}

static __global__ void fftshift2d_odd_width_odd_height(const hipFloatComplex* d_in, hipFloatComplex* d_out, int width, int height, int in_pitch, int out_pitch)
{
	// Shifting along both the width and height
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int half_width = width / 2;
	int half_height = height / 2;
	if (x < width && y < height)
	{
		if (x <= half_width)
		{
			if (y <= half_height)
			{
				int x2 = x + half_width;
				int y2 = y + half_height;
				size_t idx = (size_t)y2 * out_pitch + x2;
				size_t idx2 = (size_t)y * in_pitch + x;
				d_out[idx] = d_in[idx2];
			}
			else
			{
				int x2 = x + half_width;
				int y2 = y - half_height - 1;
				size_t idx = (size_t)y2 * out_pitch + x2;
				size_t idx2 = (size_t)y * in_pitch + x;
				d_out[idx] = d_in[idx2];
			}
		}
		else
		{
			if (y <= half_height)
			{
				int x2 = x - half_width - 1;
				int y2 = y + half_height;
				size_t idx = (size_t)y2 * out_pitch + x2;
				size_t idx2 = (size_t)y * in_pitch + x;
				d_out[idx] = d_in[idx2];
			}
			else
			{
				int x2 = x - half_width - 1;
				int y2 = y - half_height - 1;
				size_t idx = (size_t)y2 * out_pitch + x2;
				size_t idx2 = (size_t)y * in_pitch + x;
				d_out[idx] = d_in[idx2];
			}
		}
	}
}


static __global__ void fftshift2d_even_width_even_height(const hipDoubleComplex* d_in, hipDoubleComplex* d_out, int width, int height, int in_pitch, int out_pitch)
{
	// Shifting along both the width and height
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int half_width = width / 2;
	int half_height = height / 2;
	if (x < width && y < height)
	{
		int x2 = (x + half_width) % width;
		int y2 = (y + half_height) % height;
		size_t idx = (size_t)y * out_pitch + x;
		size_t idx2 = (size_t)y2 * in_pitch + x2;
		d_out[idx] = d_in[idx2];
	}
}

static __global__ void fftshift2d_odd_width_even_height(const hipDoubleComplex* d_in, hipDoubleComplex* d_out, int width, int height, int in_pitch, int out_pitch)
{
	// Shifting along both the width and height
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int half_width = width / 2;
	int half_height = height / 2;
	if (x < width && y < height)
	{
		if (x <= half_width)
		{
			int x2 = x + half_width;
			int y2 = (y + half_height) % height;
			size_t idx = (size_t)y2 * out_pitch + x2;
			size_t idx2 = (size_t)y * in_pitch + x;
			d_out[idx] = d_in[idx2];
		}
		else
		{
			int x2 = x - half_width - 1;
			int y2 = (y + half_height) % height;
			size_t idx = (size_t)y2 * out_pitch + x2;
			size_t idx2 = (size_t)y * in_pitch + x;
			d_out[idx] = d_in[idx2];
		}
	}
}

static __global__ void fftshift2d_even_width_odd_height(const hipDoubleComplex* d_in, hipDoubleComplex* d_out, int width, int height, int in_pitch, int out_pitch)
{
	// Shifting along both the width and height
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int half_width = width / 2;
	int half_height = height / 2;
	if (x < width && y < height)
	{
		if (y <= half_height)
		{
			int x2 = (x + half_width) % width;
			int y2 = y + half_height;
			size_t idx = (size_t)y2 * out_pitch + x2;
			size_t idx2 = (size_t)y * in_pitch + x;
			d_out[idx] = d_in[idx2];
		}
		else
		{
			int x2 = (x + half_width) % width;
			int y2 = y - half_height - 1;
			size_t idx = (size_t)y2 * out_pitch + x2;
			size_t idx2 = (size_t)y * in_pitch + x;
			d_out[idx] = d_in[idx2];
		}
	}
}

static __global__ void fftshift2d_odd_width_odd_height(const hipDoubleComplex* d_in, hipDoubleComplex* d_out, int width, int height, int in_pitch, int out_pitch)
{
	// Shifting along both the width and height
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int half_width = width / 2;
	int half_height = height / 2;
	if (x < width && y < height)
	{
		if (x <= half_width)
		{
			if (y <= half_height)
			{
				int x2 = x + half_width;
				int y2 = y + half_height;
				size_t idx = (size_t)y2 * out_pitch + x2;
				size_t idx2 = (size_t)y * in_pitch + x;
				d_out[idx] = d_in[idx2];
			}
			else
			{
				int x2 = x + half_width;
				int y2 = y - half_height - 1;
				size_t idx = (size_t)y2 * out_pitch + x2;
				size_t idx2 = (size_t)y * in_pitch + x;
				d_out[idx] = d_in[idx2];
			}
		}
		else
		{
			if (y <= half_height)
			{
				int x2 = x - half_width - 1;
				int y2 = y + half_height;
				size_t idx = (size_t)y2 * out_pitch + x2;
				size_t idx2 = (size_t)y * in_pitch + x;
				d_out[idx] = d_in[idx2];
			}
			else
			{
				int x2 = x - half_width - 1;
				int y2 = y - half_height - 1;
				size_t idx = (size_t)y2 * out_pitch + x2;
				size_t idx2 = (size_t)y * in_pitch + x;
				d_out[idx] = d_in[idx2];
			}
		}
	}
}




hipfftResult_t cufft::fft(const hipFloatComplex* d_in, hipFloatComplex* d_out, int N)
{
	hipfftHandle plan;
	hipfftResult result = hipfftPlan1d(&plan, N, HIPFFT_C2C, 1);

	if (result != HIPFFT_SUCCESS)
		return result;
	
	result = hipfftExecC2C(plan, (hipfftComplex*)d_in, (hipfftComplex*)d_out, HIPFFT_FORWARD);
	
	if (result != HIPFFT_SUCCESS)
	{
		hipfftDestroy(plan);
		return result;
	}

	return hipfftDestroy(plan);
}

hipfftResult_t cufft::fft(const hipDoubleComplex* d_in, hipDoubleComplex* d_out, int N)
{
	hipfftHandle plan;
	hipfftResult result = hipfftPlan1d(&plan, N, HIPFFT_Z2Z, 1);

	if (result != HIPFFT_SUCCESS)
		return result;

	result = hipfftExecZ2Z(plan, (hipfftDoubleComplex*)d_in, (hipfftDoubleComplex*)d_out, HIPFFT_FORWARD);

	if (result != HIPFFT_SUCCESS)
	{
		hipfftDestroy(plan);
		return result;
	}

	return hipfftDestroy(plan);
}

hipfftResult_t cufft::fft(const hipFloatComplex* d_in, hipFloatComplex* d_out, int width, int height, int dim, int in_pitch, int out_pitch)
{
	// 2D FFT along the dimension specified by dim, either rows or columns

	hipfftHandle plan = 0;
	hipfftResult result = HIPFFT_SUCCESS;

	int rank = 1;

	if (dim == 0)
	{
		// FFT along the width, the fastest changing dimension in memory
		// There are height rows, each of width elements
		int n[] = { width };
		int inembed[] = { width };
		int istride = 1;
		int idist = in_pitch ? (in_pitch) : (width);
		int onembed[] = { width };
		int ostride = 1;
		int odist = out_pitch ? (out_pitch) : (width);
		int batch = height;		// number of transforms to perform

		result = hipfftPlanMany(&plan, rank, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_C2C, batch);
	}
	else if (dim == 1)
	{
		int n[] = { height };
		int inembed[] = { width };
		int istride = in_pitch ? (in_pitch) : (width);
		int idist = 1;
		int onembed[] = { width };
		int ostride = out_pitch ? (out_pitch) : (width);
		int odist = 1; 
		int batch = width;		// number of transforms to perform

		result = hipfftPlanMany(&plan, rank, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_C2C, batch);
	}
	else
	{
		return HIPFFT_SETUP_FAILED;
	}

	if (result != HIPFFT_SUCCESS)
		return result;

	result = hipfftExecC2C(plan, (hipfftComplex*)d_in, (hipfftComplex*)d_out, HIPFFT_FORWARD);

	if (result != HIPFFT_SUCCESS)
	{
		fprintf(stderr, "hipfftExecC2C failed\n");
		hipfftDestroy(plan);
		return result;
	}

	return hipfftDestroy(plan);
}

hipfftResult_t cufft::fft(const hipDoubleComplex* d_in, hipDoubleComplex* d_out, int width, int height, int dim, int in_pitch, int out_pitch)
{
	// 2D FFT along the dimension specified by dim, either rows or columns

	hipfftHandle plan = 0;
	hipfftResult result = HIPFFT_SUCCESS;

	int rank = 1;

	if (dim == 0)
	{
		// FFT along the width, the fastest changing dimension in memory
		// There are height rows, each of width elements
		int n[] = { width };
		int istride = 1, ostride = 1;
		int idist = in_pitch ? (in_pitch) : (width);
		int odist = out_pitch ? (out_pitch) : (width);
		int inembed[] = { in_pitch ? (in_pitch) : (width) };
		int onembed[] = { out_pitch ? (out_pitch) : (width) };
		int batch = height;		// number of transforms to perform

		result = hipfftPlanMany(&plan, rank, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_Z2Z, batch);
	}
	else if (dim == 1)
	{
		// FFT along the height, the slowest changing dimension in memory, distance between consecutive elements is width
		// There are width columns, each of height elements

		int n[] = { height };
		int istride = in_pitch ? (in_pitch) : (width);
		int ostride = out_pitch ? (out_pitch) : (width);
		int idist = 1, odist = 1;
		int inembed[] = { in_pitch ? (in_pitch) : (width) };
		int onembed[] = { out_pitch ? (out_pitch) : (width) };
		int batch = width;		// number of transforms to perform

		result = hipfftPlanMany(&plan, rank, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_Z2Z, batch);
	}
	else
	{
		return HIPFFT_SETUP_FAILED;
	}

	if (result != HIPFFT_SUCCESS)
		return result;

	result = hipfftExecC2C(plan, (hipfftComplex*)d_in, (hipfftComplex*)d_out, HIPFFT_FORWARD);

	if (result != HIPFFT_SUCCESS)
	{
		hipfftDestroy(plan);
		return result;
	}

	return hipfftDestroy(plan);
}

hipfftResult_t cufft::fft2(const hipFloatComplex* d_in, hipFloatComplex* d_out, int width, int height, int in_pitch, int out_pitch)
{
	hipfftHandle plan = 0;
	hipfftResult result = HIPFFT_SUCCESS;

	int rank = 2;
	int n[] = { height, width };
	int istride = 1, ostride = 1;
	int idist = 1, odist = 1;
	int inembed[] = { height, in_pitch ? (in_pitch) : (width) };
	int onembed[] = { height, out_pitch ? (out_pitch) : (width) };
	int batch = 1;

	result = hipfftPlanMany(&plan, rank, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_C2C, batch);

	if (result != HIPFFT_SUCCESS)
		return result;

	result = hipfftExecC2C(plan, (hipfftComplex*)d_in, (hipfftComplex*)d_out, HIPFFT_FORWARD);

	if (result != HIPFFT_SUCCESS)
	{
		hipfftDestroy(plan);
		return result;
	}

	return hipfftDestroy(plan);
}

hipfftResult_t cufft::fft2(const hipDoubleComplex* d_in, hipDoubleComplex* d_out, int width, int height, int in_pitch, int out_pitch)
{
	hipfftHandle plan = 0;
	hipfftResult result = HIPFFT_SUCCESS;

	int rank = 2;
	int n[] = { height, width };
	int istride = 1, ostride = 1;
	int idist = 1, odist = 1;
	int inembed[] = { height, in_pitch ? (in_pitch) : (width) };
	int onembed[] = { height, out_pitch ? (out_pitch) : (width) };
	int batch = 1;

	result = hipfftPlanMany(&plan, rank, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_Z2Z, batch);

	if (result != HIPFFT_SUCCESS)
		return result;

	result = hipfftExecZ2Z(plan, (hipfftDoubleComplex*)d_in, (hipfftDoubleComplex*)d_out, HIPFFT_FORWARD);

	if (result != HIPFFT_SUCCESS)
	{
		hipfftDestroy(plan);
		return result;
	}

	return hipfftDestroy(plan);
}

hipfftResult_t cufft::ifft(const hipFloatComplex* d_in, hipFloatComplex* d_out, int N)
{
	hipfftHandle plan;
	hipfftResult result = hipfftPlan1d(&plan, N, HIPFFT_C2C, 1);
	dim3 block(256);
	dim3 grid((N + block.x - 1) / block.x);

	if (result != HIPFFT_SUCCESS)
		return result;

	result = hipfftExecC2C(plan, (hipfftComplex*)d_in, (hipfftComplex*)d_out, HIPFFT_BACKWARD);
	if (result != HIPFFT_SUCCESS)
	{
		hipfftDestroy(plan);
		return result;
	}

	result = hipfftDestroy(plan);
	if (result != HIPFFT_SUCCESS)
		return result;

	double scale = 1.0 / (double)N;

	ifft_scale_kernel << <grid, block >> > (d_out, N, scale);
	hipError_t err = hipPeekAtLastError();

	if (err != hipSuccess)
		return hipfftResult_t::HIPFFT_EXEC_FAILED;

	return hipfftResult_t::HIPFFT_SUCCESS;
}

hipfftResult_t cufft::ifft(const hipDoubleComplex* d_in, hipDoubleComplex* d_out, int N)
{
	hipfftHandle plan;
	hipfftResult result = hipfftPlan1d(&plan, N, HIPFFT_Z2Z, 1);
	dim3 block(256);
	dim3 grid((N + block.x - 1) / block.x);

	if (result != HIPFFT_SUCCESS)
		return result;

	result = hipfftExecZ2Z(plan, (hipfftDoubleComplex*)d_in, (hipfftDoubleComplex*)d_out, HIPFFT_BACKWARD);
	if (result != HIPFFT_SUCCESS)
	{
		hipfftDestroy(plan);
		return result;
	}

	result = hipfftDestroy(plan);
	if (result != HIPFFT_SUCCESS)
		return result;

	double scale = 1.0 / (double)N;
	ifft_scale_kernel << <grid, block >> > (d_out, N, scale);

	hipError_t err = hipPeekAtLastError();
	if (err != hipSuccess)
		return hipfftResult_t::HIPFFT_EXEC_FAILED;

	return hipfftResult_t::HIPFFT_SUCCESS;
}

hipfftResult_t cufft::ifft(const hipFloatComplex* d_in, hipFloatComplex* d_out, int width, int height, int dim, int in_pitch, int out_pitch)
{
	hipfftHandle plan = 0;
	hipfftResult result = HIPFFT_SUCCESS;
	hipError_t err = hipSuccess;
	int rank = 1;
	dim3 block(16, 16);
	dim3 grid((width + block.x - 1) / block.x, (height + block.y - 1) / block.y);

	if (dim == 0)
	{
		// FFT along the width, the fastest changing dimension in memory
		// There are height rows, each of width elements
		int n[] = { width };
		int inembed[] = { width };
		int istride = 1;
		int idist = in_pitch ? (in_pitch) : (width);
		int onembed[] = { width };
		int ostride = 1;
		int odist = out_pitch ? (out_pitch) : (width);
		int batch = height;		// number of transforms to perform
		result = hipfftPlanMany(&plan, rank, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_C2C, batch);
	}
	else if (dim == 1)
	{
		int n[] = { height };
		int inembed[] = { width };
		int istride = in_pitch ? (in_pitch) : (width);
		int idist = 1;
		int onembed[] = { width };
		int ostride = out_pitch ? (out_pitch) : (width);
		int odist = 1;
		int batch = width;		// number of transforms to perform
		result = hipfftPlanMany(&plan, rank, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_C2C, batch);
	}
	else
	{
		return HIPFFT_SETUP_FAILED;
	}

	if (result != HIPFFT_SUCCESS)
		return result;

	result = hipfftExecC2C(plan, (hipfftComplex*)d_in, (hipfftComplex*)d_out, HIPFFT_BACKWARD);
	if (result != HIPFFT_SUCCESS)
	{
		hipfftDestroy(plan);
		return result;
	}

	result = hipfftDestroy(plan);
	if (result != HIPFFT_SUCCESS)
		return result;

	double scale = dim == 0 ? 1.0 / (double)width : 1.0 / (double)height;
	//double scale = 1.0;
	ifft2_scale_kernel << <grid, block >> > (d_out, width, height, scale, out_pitch ? (out_pitch) : width);
	err = hipPeekAtLastError();

	if (err != hipSuccess)
		return hipfftResult_t::HIPFFT_EXEC_FAILED;

	return hipfftResult_t::HIPFFT_SUCCESS;
}

hipfftResult_t cufft::ifft(const hipDoubleComplex* d_in, hipDoubleComplex* d_out, int width, int height, int dim, int in_pitch, int out_pitch)
{
	hipfftHandle plan = 0;
	hipfftResult result = HIPFFT_SUCCESS;
	hipError_t err = hipSuccess;
	int rank = 1;
	dim3 block(16, 16);
	dim3 grid((width + block.x - 1) / block.x, (height + block.y - 1) / block.y);

	if (dim == 0)
	{
		// FFT along the width, the fastest changing dimension in memory
		// There are height rows, each of width elements
		int n[] = { width };
		int inembed[] = { width };
		int istride = 1;
		int idist = in_pitch ? (in_pitch) : (width);
		int onembed[] = { width };
		int ostride = 1;
		int odist = out_pitch ? (out_pitch) : (width);
		int batch = height;		// number of transforms to perform
		result = hipfftPlanMany(&plan, rank, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_Z2Z, batch);
	}
	else if (dim == 1)
	{
		int n[] = { height };
		int inembed[] = { width };
		int istride = in_pitch ? (in_pitch) : (width);
		int idist = 1;
		int onembed[] = { width };
		int ostride = out_pitch ? (out_pitch) : (width);
		int odist = 1;
		int batch = width;		// number of transforms to perform
		result = hipfftPlanMany(&plan, rank, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_Z2Z, batch);
	}
	else
	{
		return HIPFFT_SETUP_FAILED;
	}

	if (result != HIPFFT_SUCCESS)
		return result;

	result = hipfftExecZ2Z(plan, (hipDoubleComplex*)d_in, (hipDoubleComplex*)d_out, HIPFFT_BACKWARD);
	if (result != HIPFFT_SUCCESS)
	{
		hipfftDestroy(plan);
		return result;
	}

	result = hipfftDestroy(plan);
	if (result != HIPFFT_SUCCESS)
		return result;

	double scale = dim == 0 ? 1.0 / (double)width : 1.0 / (double)height;
	//double scale = 1.0;
	ifft2_scale_kernel << <grid, block >> > (d_out, width, height, scale, out_pitch ? (out_pitch) : width);
	err = hipPeekAtLastError();

	if (err != hipSuccess)
		return hipfftResult_t::HIPFFT_EXEC_FAILED;

	return hipfftResult_t::HIPFFT_SUCCESS;
}

hipfftResult_t cufft::ifft2(const hipFloatComplex* d_in, hipFloatComplex* d_out, int width, int height, int in_pitch, int out_pitch)
{
	hipfftHandle plan = 0;
	hipfftResult result = HIPFFT_SUCCESS;
	hipError_t err = hipSuccess;

	dim3 block(16, 16);
	dim3 grid((width + block.x - 1) / block.x, (height + block.y - 1) / block.y);

	int rank = 2;
	int n[] = { height, width };
	int istride = 1, ostride = 1;
	int idist = 1, odist = 1;
	int inembed[] = { height, in_pitch ? (in_pitch) : (width) };
	int onembed[] = { height, out_pitch ? (out_pitch) : (width) };
	int batch = 1;

	result = hipfftPlanMany(&plan, rank, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_C2C, batch);

	if (result != HIPFFT_SUCCESS)
		return result;

	result = hipfftExecC2C(plan, (hipfftComplex*)d_in, (hipfftComplex*)d_out, HIPFFT_BACKWARD);

	if (result != HIPFFT_SUCCESS)
	{
		hipfftDestroy(plan);
		return result;
	}

	result = hipfftDestroy(plan);

	if (result != HIPFFT_SUCCESS)
		return result;

	double scale = 1.0 / (double)(width * height);

	ifft2_scale_kernel << <grid, block >> > (d_out, width, height, scale, out_pitch ? (out_pitch) : width);
	err = hipPeekAtLastError();

	if (err != hipSuccess)
		return hipfftResult_t::HIPFFT_EXEC_FAILED;

	return hipfftResult_t::HIPFFT_SUCCESS;
}

hipfftResult_t cufft::ifft2(const hipDoubleComplex* d_in, hipDoubleComplex* d_out, int width, int height, int in_pitch, int out_pitch)
{
	hipfftHandle plan = 0;
	hipfftResult result = HIPFFT_SUCCESS;
	hipError_t err = hipSuccess;

	dim3 block(16, 16);
	dim3 grid((width + block.x - 1) / block.x, (height + block.y - 1) / block.y);

	int rank = 2;
	int n[] = { height, width };
	int istride = 1, ostride = 1;
	int idist = 1, odist = 1;
	int inembed[] = { height, in_pitch ? (in_pitch) : (width) };
	int onembed[] = { height, out_pitch ? (out_pitch) : (width) };
	int batch = 1;

	result = hipfftPlanMany(&plan, rank, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_Z2Z, batch);

	if (result != HIPFFT_SUCCESS)
		return result;

	result = hipfftExecZ2Z(plan, (hipfftDoubleComplex*)d_in, (hipfftDoubleComplex*)d_out, HIPFFT_BACKWARD);

	if (result != HIPFFT_SUCCESS)
	{
		hipfftDestroy(plan);
		return result;
	}

	result = hipfftDestroy(plan);
	if (result != HIPFFT_SUCCESS)
		return result;

	double scale = 1.0 / (double)(width * height);

	ifft2_scale_kernel << <grid, block >> > (d_out, width, height, scale, out_pitch ? (out_pitch) : width);
	err = hipPeekAtLastError();

	if (err != hipSuccess)
		return hipfftResult_t::HIPFFT_EXEC_FAILED;

	return hipfftResult_t::HIPFFT_SUCCESS;
}

hipfftResult_t cufft::fftshift(const hipFloatComplex* d_in, hipFloatComplex* d_out, int N)
{
	dim3 block(256);
	dim3 grid((N + block.x - 1) / block.x);

	// Check to see if pointers overlap at all, if they do, just print a warning statement
	// The results are undefined, but the program shouldnt crash, unless N is out of bounds
	uintptr_t in_ptr = (uintptr_t)d_in;
	uintptr_t out_ptr = (uintptr_t)d_out;
	uintptr_t in_end = in_ptr + N * sizeof(hipFloatComplex);

	if (in_ptr <= out_ptr && out_ptr < in_end)
	{
		fprintf(stderr, "Warning: fftshift: input and output pointers overlap\n");
	}

	if (N % 2 == 0)
		fftshift_even_kernel << <grid, block >> > (d_in, d_out, N);
	else
		fftshift_odd_kernel << <grid, block >> > (d_in, d_out, N);

	hipError_t err = hipPeekAtLastError();
	if (err != hipSuccess)
		return hipfftResult_t::HIPFFT_EXEC_FAILED;
	
	return hipfftResult_t::HIPFFT_SUCCESS;
}

hipfftResult_t cufft::fftshift(const hipDoubleComplex* d_in, hipDoubleComplex* d_out, int N)
{
	dim3 block(256);
	dim3 grid((N + block.x - 1) / block.x);

	// Check to see if pointers overlap at all, if they do, just print a warning statement
	// The results are undefined, but the program shouldnt crash, unless N is out of bounds
	uintptr_t in_ptr = (uintptr_t)d_in;
	uintptr_t out_ptr = (uintptr_t)d_out;
	uintptr_t in_end = in_ptr + N * sizeof(hipFloatComplex);

	if (in_ptr <= out_ptr && out_ptr < in_end)
	{
		fprintf(stderr, "Warning: fftshift: input and output pointers overlap\n");
	}

	if (N % 2 == 0)
		fftshift_even_kernel << <grid, block >> > (d_in, d_out, N);
	else
		fftshift_odd_kernel << <grid, block >> > (d_in, d_out, N);

	hipError_t err = hipPeekAtLastError();
	if (err != hipSuccess)
		return hipfftResult_t::HIPFFT_EXEC_FAILED;

	return hipfftResult_t::HIPFFT_SUCCESS;
}

hipfftResult_t cufft::fftshift(const hipFloatComplex* d_in, hipFloatComplex* d_out, int width, int height, int dim, int in_pitch, int out_pitch)
{
	dim3 block(16, 16);
	dim3 grid((width + block.x - 1) / block.x, (height + block.y - 1) / block.y);

	// Check to see if pointers overlap at all, if they do, just print a warning statement
	// The results are undefined, but the program shouldnt crash, unless N is out of bounds
	uintptr_t in_ptr = (uintptr_t)d_in;
	uintptr_t out_ptr = (uintptr_t)d_out;
	uintptr_t in_end = in_ptr + height * in_pitch * sizeof(hipFloatComplex);

	if (in_ptr <= out_ptr && out_ptr < in_end)
	{
		fprintf(stderr, "Warning: fftshift: input and output pointers overlap\n");
	}

	in_pitch = in_pitch ? in_pitch : width;
	out_pitch = out_pitch ? out_pitch : width;

	if (dim == 0)
	{
		if (width % 2 == 0)
			fftshift_even_kernel_width << <grid, block >> > (d_in, d_out, width, height, in_pitch, out_pitch);
		else
			fftshift_odd_kernel_width << <grid, block >> > (d_in, d_out, width, height, in_pitch, out_pitch);
	}
	else if (dim == 1)
	{
		if (height % 2 == 0)
			fftshift_even_kernel_height << <grid, block >> > (d_in, d_out, width, height, in_pitch, out_pitch);
		else
			fftshift_odd_kernel_height << <grid, block >> > (d_in, d_out, width, height, in_pitch, out_pitch);
	}
	else
	{
		return hipfftResult_t::HIPFFT_SETUP_FAILED;
	}

	hipError_t err = hipPeekAtLastError();
	if (err != hipSuccess)
		return hipfftResult_t::HIPFFT_EXEC_FAILED;

	return hipfftResult_t::HIPFFT_SUCCESS;

}

hipfftResult_t cufft::fftshift(const hipDoubleComplex* d_in, hipDoubleComplex* d_out, int width, int height, int dim, int in_pitch, int out_pitch)
{
	dim3 block(16, 16);
	dim3 grid((width + block.x - 1) / block.x, (height + block.y - 1) / block.y);

	// Check to see if pointers overlap at all, if they do, just print a warning statement
	// The results are undefined, but the program shouldnt crash, unless N is out of bounds
	uintptr_t in_ptr = (uintptr_t)d_in;
	uintptr_t out_ptr = (uintptr_t)d_out;
	uintptr_t in_end = in_ptr + height * in_pitch * sizeof(hipDoubleComplex);

	if (in_ptr <= out_ptr && out_ptr < in_end)
	{
		fprintf(stderr, "Warning: fftshift: input and output pointers overlap\n");
	}

	in_pitch = in_pitch ? in_pitch : width;
	out_pitch = out_pitch ? out_pitch : width;

	if (dim == 0)
	{
		if (width % 2 == 0)
			fftshift_even_kernel_width << <grid, block >> > (d_in, d_out, width, height, in_pitch, out_pitch);
		else
			fftshift_odd_kernel_width << <grid, block >> > (d_in, d_out, width, height, in_pitch, out_pitch);
	}
	else if (dim == 1)
	{
		if (height % 2 == 0)
			fftshift_even_kernel_height << <grid, block >> > (d_in, d_out, width, height, in_pitch, out_pitch);
		else
			fftshift_odd_kernel_height << <grid, block >> > (d_in, d_out, width, height, in_pitch, out_pitch);
	}
	else
	{
		return hipfftResult_t::HIPFFT_SETUP_FAILED;
	}

	hipError_t err = hipPeekAtLastError();
	if (err != hipSuccess)
		return hipfftResult_t::HIPFFT_EXEC_FAILED;

	return hipfftResult_t::HIPFFT_SUCCESS;
}

hipfftResult_t cufft::fftshift2(const hipFloatComplex* d_in, hipFloatComplex* d_out, int width, int height, int in_pitch, int out_pitch)
{
	dim3 block(16, 16);
	dim3 grid((width + block.x - 1) / block.x, (height + block.y - 1) / block.y);

	// Check to see if pointers overlap at all, if they do, just print a warning statement
	// The results are undefined, but the program shouldnt crash, unless N is out of bounds
	uintptr_t in_ptr = (uintptr_t)d_in;
	uintptr_t out_ptr = (uintptr_t)d_out;
	uintptr_t in_end = in_ptr + height * in_pitch * sizeof(hipFloatComplex);

	if (in_ptr <= out_ptr && out_ptr < in_end)
	{
		fprintf(stderr, "Warning: fftshift: input and output pointers overlap\n");
	}

	in_pitch = in_pitch ? in_pitch : width;
	out_pitch = out_pitch ? out_pitch : width;

	if (width % 2 == 0)
	{
		if (height % 2 == 0)
			fftshift2d_even_width_even_height << <grid, block >> > (d_in, d_out, width, height, in_pitch, out_pitch);
		else
			fftshift2d_even_width_odd_height << <grid, block >> > (d_in, d_out, width, height, in_pitch, out_pitch);
	}
	else
	{
		if (height % 2 == 0)
			fftshift2d_odd_width_even_height << <grid, block >> > (d_in, d_out, width, height, in_pitch, out_pitch);
		else
			fftshift2d_odd_width_odd_height << <grid, block >> > (d_in, d_out, width, height, in_pitch, out_pitch);
	}

	hipError_t err = hipPeekAtLastError();
	if (err != hipSuccess)
		return hipfftResult_t::HIPFFT_EXEC_FAILED;

	return hipfftResult_t::HIPFFT_SUCCESS;
}

hipfftResult_t cufft::fftshift2(const hipDoubleComplex* d_in, hipDoubleComplex* d_out, int width, int height, int in_pitch, int out_pitch)
{
	dim3 block(16, 16);
	dim3 grid((width + block.x - 1) / block.x, (height + block.y - 1) / block.y);

	// Check to see if pointers overlap at all, if they do, just print a warning statement
	// The results are undefined, but the program shouldnt crash, unless N is out of bounds
	uintptr_t in_ptr = (uintptr_t)d_in;
	uintptr_t out_ptr = (uintptr_t)d_out;
	uintptr_t in_end = in_ptr + height * in_pitch * sizeof(hipDoubleComplex);

	if (in_ptr <= out_ptr && out_ptr < in_end)
	{
		fprintf(stderr, "Warning: fftshift: input and output pointers overlap\n");
	}

	in_pitch = in_pitch ? in_pitch : width;
	out_pitch = out_pitch ? out_pitch : width;

	if (width % 2 == 0)
	{
		if (height % 2 == 0)
			fftshift2d_even_width_even_height << <grid, block >> > (d_in, d_out, width, height, in_pitch, out_pitch);
		else
			fftshift2d_even_width_odd_height << <grid, block >> > (d_in, d_out, width, height, in_pitch, out_pitch);
	}
	else
	{
		if (height % 2 == 0)
			fftshift2d_odd_width_even_height << <grid, block >> > (d_in, d_out, width, height, in_pitch, out_pitch);
		else
			fftshift2d_odd_width_odd_height << <grid, block >> > (d_in, d_out, width, height, in_pitch, out_pitch);
	}

	hipError_t err = hipPeekAtLastError();
	if (err != hipSuccess)
		return hipfftResult_t::HIPFFT_EXEC_FAILED;

	return hipfftResult_t::HIPFFT_SUCCESS;
}

